#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include<math.h>
// the kernal
__global__ void RankSortParallel(int*input, int*output ,int N){
    // the index which the thread will work on
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index < N) {
        // variable to save the rank of af the current thread cell
        int rank = 0;
        int i;
        for (i = 0; i < N; i++) {
            // if the cell is before the index cell and the value in that cell <= the value in the thread index cell then rank++
            if (i < index) {
                if (input[i] <= input[index]) {
                    rank++;
                }
            }
            // if the cell is after the index cell and the value in that cell < the value in the thread index cell then rank++
            else if (i > index) {
                if (input[i] < input[index]) {
                    rank++;
                }
            }
        }
    // put the value in its proper location(the rank)
    output[rank] = input[index];
    }
}

int main(){

    // initlize the data and some variables
    int data[] = {8,16,2,4,-5,1,6,-16,20,-16,10};
    int size = sizeof(data);
    int numOfelementsInInput = size / sizeof(int);


    int i;
    printf("the input data:\n");
    for (i = 0; i < numOfelementsInInput; i++) {
        printf("%d,", data[i]);
    }
    
    // create an array for input data in the gpu memory
    int* input_gpu= NULL;
    hipMalloc((void**)&input_gpu,size);

    // create an array to save the data in cpu(host) after executing the kernal
    int* output = (int*)malloc(size);

    // create an array for output data in the gpu memory
    int* output_gpu = NULL;
    hipMalloc((void**)&output_gpu, size);
    
    
    // cupy data from the data array(in host) to the input array in the gpu
    hipMemcpy(input_gpu, data, size, hipMemcpyHostToDevice);
    
    //run the kernal with an array of blocks, each block has an array of threads    
    float threadsPerBlock = 512.0;
    RankSortParallel<<<ceil(numOfelementsInInput/threadsPerBlock),threadsPerBlock>>>(input_gpu, output_gpu, numOfelementsInInput);
    
    // wait the kernal to finish executing
    hipDeviceSynchronize();

    // copy the results back to cpu
    hipMemcpy(output, output_gpu, size, hipMemcpyDeviceToHost);
    
    // print the output
    printf("\nThe sorted array:\n");
    for (i = 0; i < numOfelementsInInput; i++) {
        printf("%d,", output[i]);
    }
}
